
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, hostreal _value)
{
  thrust::device_ptr<real> self_data(self->data);
  real value = hostrealToReal(_value);
  thrust::fill(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+self->size, value);
}

void THCStorage_(resize)(THCState *state, THCStorage *self, long size)
{
  THArgCheck(size >= 0, 2, "invalid size");

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    THError("Trying to resize storage that is not resizable");

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM) {
      THCnmemCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(real));
    }
    self->data = NULL;
    self->size = 0;
  }
  else
  {
    real *data = NULL;
    // update heap *before* attempting malloc, to free space for the malloc
    THCHeapUpdate(state, size * sizeof(real));
    cnmemStatus_t status = THCudaMalloc(state, (void**)(&data), size * sizeof(real));
    if(status != CNMEM_STATUS_SUCCESS) {
      THCHeapUpdate(state, -size * sizeof(real));
    }
    THCnmemCheck(status);

    if (self->data) {
      THCudaCheck(cudaMemcpyAsync(data,
                                  self->data,
                                  THMin(self->size, size) * sizeof(real),
                                  cudaMemcpyDeviceToDevice,
                                  THCState_getCurrentStream(state)));
      THCnmemCheck(THCudaFree(state, self->data));
      THCHeapUpdate(state, -self->size * sizeof(real));
    }

    self->data = data;
    self->size = size;
  }
}
#endif
